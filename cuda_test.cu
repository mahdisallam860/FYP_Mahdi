#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int count;
    hipGetDeviceCount(&count);
    std::cout << "CUDA Device Count: " << count << std::endl;
    return 0;
}
